#include "hip/hip_runtime.h"
// Raphael Garnier

#include <iostream>
#include <fstream>
#include <complex>
#include <chrono>
#include <hip/hip_complex.h>

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#define RESOLUTION 2000 # resolution of the output image

#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)
#define N (HEIGHT * WIDTH)

#define STEP ((double)RATIO_X / WIDTH)

#define DEGREE 2        // Degree of the polynomial
#define ITERATIONS 5000 // Maximum number of iterations on a pixel

using namespace std;


__global__ void GPUFunction(int *imageGPU){
        
    int pos = blockIdx.x * blockDim.x + threadIdx.x; // get the number of the pixel

    if (pos < WIDTH * HEIGHT) // compute only if the pixel number is less than the total of pixels
    {
        imageGPU[pos] = 0; // initialize the value of the pixel

        const int row = pos / WIDTH;
        const int col = pos % WIDTH;

        const hipDoubleComplex cc = make_hipDoubleComplex(col * STEP + MIN_X, row * STEP + MIN_Y);
        
        hipDoubleComplex zz = make_hipDoubleComplex(0,0);
        
        for (int i = 1; i <= ITERATIONS; i++)
        {
            zz = hipCmul(zz, zz); // zz is squared
            zz = hipCadd(zz, cc); // zz is added to cc

            // If it is not convergent
            if (hipCabs(zz) >= 2)
            {
                imageGPU[pos] = i; // store the value
                break;
            }
        }
    }
}


int main(int argc, char **argv)
{
    int* imageGPU;
    dim3 threads(32);
    dim3 blocks((N+threads.x-1)/threads.x);
    const auto start = chrono::steady_clock::now();

    int *const imageCPU = new int[N]; // array storing the pixels on the CPU

    hipMalloc( (void**)&imageGPU, N*sizeof(int) );  // malloc another array on the GPU

    GPUFunction<<<blocks, threads>>>(imageGPU);  // launch the computation on the GPU

    hipMemcpy(imageCPU, imageGPU, N*sizeof(int), hipMemcpyDeviceToHost); // copy data from the GPU to the CPU
    hipDeviceSynchronize();

    const auto end = chrono::steady_clock::now();
    cout << "Time elapsed: "
         << chrono::duration_cast<chrono::seconds>(end - start).count()
         << " seconds." << endl;

    // Write the result to a file
    ofstream matrix_out;

    if (argc < 2)
    {
        cout << "Please specify the output file as a parameter." << endl;
        return -1;
    }

    matrix_out.open(argv[1], ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << imageCPU[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    delete[] imageCPU; // It's here for coding style, but useless
    hipFree(imageGPU); // same, but for cuda
    return 0;
}
